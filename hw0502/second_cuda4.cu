// Second CUDA program
// Ping-Che Chen


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 256
#define BLOCK_SIZE 32

__global__ static void matMultCUDA(const float* a, size_t lda, const float* b, size_t ldb, float* c, size_t ldc, int n)
{
	extern __shared__ float data[];
	const int tid = threadIdx.x;
	const int row = blockIdx.x;
	int i,j;
	
	for(i = tid; i < n; i += blockDim.x){
		data[i] = a[row * lda + i];
	}
	__syncthreads();

    for(j = tid; j < n; j += blockDim.x)
	{
		float t=0;
		float y=0;
		for(i = 0; i < n; i++)
		{
			float r;
			y -= data[i] * b[i * ldb + j];
			r = t - y;
			y = (r - t) + y;
			t = r;
		}
		c[row * ldc + j] = t;
	}
}

clock_t matmultCUDA(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n)
{
	float *ac, *bc, *cc;
	clock_t start, end;
	size_t pitch_a, pitch_b, pitch_c;
	
	start = clock();
	hipMallocPitch((void**) &ac, &pitch_a, sizeof(float) * n, n);
	hipMallocPitch((void**) &bc, &pitch_b, sizeof(float) * n, n);
	hipMallocPitch((void**) &cc, &pitch_c, sizeof(float) * n, n);

	hipMemcpy2D(ac, pitch_a, a, sizeof(float) * lda, sizeof(float) * n, n, hipMemcpyHostToDevice);
	hipMemcpy2D(bc, pitch_b, b, sizeof(float) * ldb, sizeof(float) * n, n, hipMemcpyHostToDevice);
	matMultCUDA<<<n, NUM_THREADS, sizeof(float) * n>>>
        (ac, pitch_a / sizeof(float), bc, pitch_b / sizeof(float),
        cc, pitch_c / sizeof(float), n);

	//int blocks = (n + NUM_THREADS - 1) / NUM_THREADS;
	//matMultCUDA<<<blocks * n, NUM_THREADS>>> (ac, n, bc, n, cc, n, n);
	//matMultCUDA<<<n, NUM_THREADS, sizeof(float) * n>>>(ac, n, bc, n, cc, n, n);

	//cudaMemcpy2D(c, sizeof(float)*ldc, cc, sizeof(float) * n,  sizeof(float) * n, n, cudaMemcpyDeviceToHost);
	//cudaMemcpy2D(c, sizeof(float) * ldc, cc, sizeof(float) * n, sizeof(float) *n, n, cudaMemcpyDeviceToHost);
	hipMemcpy2D(c, sizeof(float) * ldc, cc, pitch_c,
        sizeof(float) * n, n, hipMemcpyDeviceToHost);

	hipFree(ac);
	hipFree(bc);
	hipFree(cc);
	end = clock();
	return end-start;
}


void matmult(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n)
{
	int i, j, k;

	for(i = 0; i < n; i++) {
		for(j = 0; j < n; j++) {
			double t = 0;
			for(k = 0; k < n; k++) {
				t += a[i * lda + k] * b[k * ldb + j];
			}
			c[i * ldc + j] = t;
		}
	}
}


void matgen(float* a, int lda, int n)
{
	int i, j;

	for(i = 0; i < n; i++) {
		for(j = 0; j < n; j++) 
		{
			a[i * lda + j] = (float) rand() / RAND_MAX + (float) rand() / (RAND_MAX * RAND_MAX);
		}
	}
}


void compare_mat(const float* a, int lda, const float* b, int ldb, int n)
{
	float max_err = 0;
	float average_err = 0;
	int i, j;

	for(i = 0; i < n; i++) {
		for(j = 0; j < n; j++) {
			if(b[i * ldb + j] != 0) {
				float err = fabs((a[i * lda + j] - b[i * ldb + j]) / b[i * ldb + j]);
				if(max_err < err) max_err = err;
				average_err += err;
			}
		}
	}

	printf("Max error: %g  Average error: %g\n", max_err, average_err / (n * n));
}


bool InitCUDA()
{
	int count;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}

	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}


int main()
{
	float *a, *b, *c, *d;
	int n = 1000;

	if(!InitCUDA()) {
		return 0;
	}

	a = (float*) malloc(sizeof(float) * n * n);
	b = (float*) malloc(sizeof(float) * n * n);
	c = (float*) malloc(sizeof(float) * n * n);
	d = (float*) malloc(sizeof(float) * n * n);

	srand(0);

	matgen(a, n, n);
	matgen(b, n, n);

	clock_t time = matmultCUDA(a, n, b, n, c, n, n);

	matmult(a, n, b, n, d, n, n);
	compare_mat(c, n, d, n, n);

	double sec = (double) time / CLOCKS_PER_SEC;
	printf("Time used: %.4lf   (%.2lf GFLOPS)\n", sec, 2.0 * n * n * n / (sec * 1E9));

	free(a);
	free(b);
	free(c);
	free(d);

	return 0;
}
